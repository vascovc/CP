
#include <hip/hip_runtime.h>
#include  <stdio.h> 
#include  <time.h> 

#define  SIZE (16) 

//  Kernel definition, see also section 2.1 of NVIDIA CUDA Programming Guide 
__global__  void arrAdd(float *A, float *B, float *C) 
{ 
    // threadIdx.x is a built-in variable provided by CUDA at runtime 
    // It represents the thread index inside the block

    int id = threadIdx.x; // id: unique thread identifier

    C[id] = A[id] + B[id]; 
} 

int  main(void) 
{ 
    float A[SIZE], B[SIZE], C[SIZE]; 
    float *devPtrA; 
    float *devPtrB; 
    float *devPtrC; 
    int memsize = SIZE * sizeof(float); 

    // Initialize arrays
    srand (time(NULL));
    for(int i=0; i < SIZE; i++) 
    {
        A[i]=rand() % 100;
        B[i]=rand() % 100;
    }

    hipSetDevice(1);    // Select GPU device (can be 0 to 1)

    // Allocate device memory for A, B and C arrays
    hipMalloc((void**)&devPtrA, memsize); 
    hipMalloc((void**)&devPtrB, memsize); 
    hipMalloc((void**)&devPtrC, memsize); 

    // Copy data (data to process) from host to device (from CPU to GPU)
    hipMemcpy(devPtrA, A, memsize,  hipMemcpyHostToDevice); 
    hipMemcpy(devPtrB, B, memsize,  hipMemcpyHostToDevice); 

    // Execute the Kernel 
    arrAdd <<<1, SIZE>>> (devPtrA,  devPtrB, devPtrC); // launch 1 block with SIZE threads

    // Copy data from device (results) back to host 
    hipMemcpy(C, devPtrC, memsize,  hipMemcpyDeviceToHost); 

    // Show results
    printf("     A      B       C\n");
    for (int i=0; i < SIZE; i++) 
    {
        printf("%2d: %4.1f + %4.1f = %5.1f\n", i, A[i], B[i], C[i]); 
    }

    // Free device memory
    hipFree(devPtrA); 
    hipFree(devPtrB); 
    hipFree(devPtrC); 
} 

