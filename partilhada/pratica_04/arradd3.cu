
#include <hip/hip_runtime.h>
#include  <stdio.h> 
#include  <time.h> 

#define  SIZE (16) 

//  Kernel definition, see also section 2.1 of NVIDIA CUDA Programming Guide 
__global__  void arrAdd(float *A, float *B, float *C) 
{ 
    // TODO: determine id
    int id;

    id = threadIdx.x + threadIdx.y*blockDim.x + blockIdx.x*blockDim.x*blockDim.y ;

    if(id < SIZE)
    {
        C[id] = A[id] + B[id]; 
    }
} 
 
int  main(void) 
{ 
    float A[SIZE], B[SIZE], C[SIZE]; 
    float *devPtrA; 
    float *devPtrB; 
    float *devPtrC; 
    int memsize = SIZE * sizeof(float); 

    // Initialize arrays
    srand (time(NULL));
    for(int i=0; i < SIZE; i++) 
    {
        A[i]=rand() % 100;
        B[i]=rand() % 100;
    }

    hipSetDevice(1);    // Select GPU device (can be 0 to 1)

    // Allocate device memory for A, B and C arrays
    hipMalloc((void**)&devPtrA, memsize); 
    hipMalloc((void**)&devPtrB, memsize); 
    hipMalloc((void**)&devPtrC, memsize); 

    // Copy data (data to process) from host to device (from CPU to GPU)
    hipMemcpy(devPtrA, A, memsize,  hipMemcpyHostToDevice); 
    hipMemcpy(devPtrB, B, memsize,  hipMemcpyHostToDevice); 

    // __global__ functions are called:  Func <<< dim grid, dim block >>> (parameter); 
//    dim3 dimGrid(SIZE/(blockDim.x*blockDim.y));
    dim3 dimBlock(2,2);
    dim3 dimGrid(SIZE/(2*2));

    // Execute the Kernel
    arrAdd <<<dimGrid, dimBlock>>> (devPtrA,  devPtrB, devPtrC); 

    // Copy data from device (results) back to host 
    hipMemcpy(C, devPtrC, memsize,  hipMemcpyDeviceToHost); 

    // Show results
    printf("     A      B       C\n");
    for (int i=0; i < SIZE; i++) 
    {
        printf("%2d: %4.1f + %4.1f = %5.1f\n", i, A[i], B[i], C[i]); 
    }

    // Free device memory
    hipFree(devPtrA); 
    hipFree(devPtrB); 
    hipFree(devPtrC); 
} 


