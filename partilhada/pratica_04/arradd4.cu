
#include <hip/hip_runtime.h>
#include  <stdio.h> 
#include  <time.h> 

#define  NROWS (4) 
#define  NCOLS (4) 
#define  SIZE (NROWS*NCOLS) 


int compare(float *a1, float *a2);


//  Kernel definition, see also section 2.1 of NVIDIA CUDA Programming Guide 
__global__  void arrAdd(float *A, float *B, float *C) 
{ 
    // TODO: determine id
    int id;

    id = (threadIdx.x + blockIdx.x*blockDim.x)+NCOLS*(blockIdx.y*blockDim.y+threadIdx.y);

    if(id < SIZE)
    {
        C[id] = A[id] + B[id]; 
    }
} 

int  main(void) 
{ 
    float A[SIZE], B[SIZE], D[SIZE], H[SIZE];
    float *devPtrA; 
    float *devPtrB; 
    float *devPtrD; 
    int memsize = SIZE * sizeof(float); 
    float devExecTime;

    hipSetDevice(1);    // Select GPU device (can be 0 to 1)

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Initialize arrays
    srand (time(NULL));
    for(int i=0; i < SIZE; i++) 
    {
        A[i]=rand() % 100;
        B[i]=rand() % 100;
    }

    printf("Starting HOST...\n");

    for(int i=0; i < NCOLS; i++)
    {
        for(int j=0; j < NROWS; j++)
        {
            int id = j + i * NROWS;
            H[id] = A[id] + B[id];
        }
    }

    // Allocate device memory for A, B and D arrays
    hipMalloc((void**)&devPtrA, memsize); 
    hipMalloc((void**)&devPtrB, memsize); 
    hipMalloc((void**)&devPtrD, memsize); 

    printf("Starting DEVICE...\n");
    hipEventRecord(start);

    // Copy data (data to process) from host to device (from CPU to GPU)
    hipMemcpy(devPtrA, A, memsize,  hipMemcpyHostToDevice); 
    hipMemcpy(devPtrB, B, memsize,  hipMemcpyHostToDevice); 

    // __global__ functions are called:  Func <<< dim grid, dim block >>> (parameter); 
    dim3 dimBlock(2,2);
    dim3 dimGrid((NROWS+dimBlock.x-1)/dimBlock.x,(NCOLS+dimBlock.y-1)/dimBlock.y);

    // Execute the Kernel 
    arrAdd <<<dimGrid, dimBlock>>> (devPtrA,  devPtrB, devPtrD); 

    // Copy data from device (results) back to host 
    hipMemcpy(D, devPtrD, memsize,  hipMemcpyDeviceToHost); 

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&devExecTime, start, stop); //Exec time = elapsed time

    // Show results
    printf("     A      B       D      H\n");
    for (int i=0; i < SIZE; i++) 
    {
        printf("%2d: %4.1f + %4.1f = %5.1f [%5.1f]\n", i, A[i], B[i], D[i], H[i]); 
    }

    printf("\nOutput arrays (H/D) are %s\n", compare(D, H) == 1 ? "EQUAL" : "DIFFERENT");

    printf("\nDevice execution time [ms]: %7.4f\n", devExecTime);

    // Free device memory
    hipFree(devPtrA); 
    hipFree(devPtrB); 
    hipFree(devPtrD); 
} 

int compare(float *a1, float *a2)
{
    int i, j, equal = 1;
    for(j=0; (j < NROWS) && equal; j++)
    {
        for(i=0; (i < NCOLS) && equal; i++)
        {
            int id = i + j * NCOLS;
            if(a1[id] != a2[id])
                equal = 0;
        }
    }
    return equal;
}


