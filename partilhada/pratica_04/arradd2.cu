
#include <hip/hip_runtime.h>
#include  <stdio.h> 
#include  <time.h> 

#define  SIZE (16) 

//  Kernel definition, see also section 2.1 of NVIDIA CUDA Programming Guide 
__global__  void arrAdd(float *A, float *B, float *C) 
{ 
    // TODO: determine id
    int id;

    id = threadIdx.x + blockIdx.x*blockDim.x;

    C[id] = A[id] + B[id]; 
} 

int  main(void) 
{ 
    float A[SIZE], B[SIZE], C[SIZE]; 
    float *devPtrA; 
    float *devPtrB; 
    float *devPtrC; 
    int memsize = SIZE * sizeof(float); 

    // Initialize arrays
    srand (time(NULL));
    for(int i=0; i < SIZE; i++) 
    {
        A[i]=rand() % 100;
        B[i]=rand() % 100;
    }

    hipSetDevice(1);    // Select GPU device (can be 0 to 1)

    // Allocate device memory for A, B and C arrays
    hipMalloc((void**)&devPtrA, memsize); 
    hipMalloc((void**)&devPtrB, memsize); 
    hipMalloc((void**)&devPtrC, memsize); 

    // Copy data (data to process) from host to device (from CPU to GPU)
    hipMemcpy(devPtrA, A, memsize,  hipMemcpyHostToDevice); 
    hipMemcpy(devPtrB, B, memsize,  hipMemcpyHostToDevice); 

    // Execute the Kernel
    // TO DO: Determine the number of blocks
    int threads_per_block = 4;
    arrAdd <<<SIZE/threads_per_block, threads_per_block>>> (devPtrA,  devPtrB, devPtrC); // launch ??? blocks with 2 threads each

    // Copy data from device (results) back to host 
    hipMemcpy(C, devPtrC, memsize,  hipMemcpyDeviceToHost); 

    // Show results
    printf("     A      B       C\n");
    for (int i=0; i < SIZE; i++) 
    {
        printf("%2d: %4.1f + %4.1f = %5.1f\n", i, A[i], B[i], C[i]); 
    }

    // Free device memory
    hipFree(devPtrA); 
    hipFree(devPtrB); 
    hipFree(devPtrC); 
} 

