#include "hip/hip_runtime.h"

// Based on CUDA SDK template from NVIDIA

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <assert.h>
#include <float.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_image.h>

#define max(a,b) (((a)>(b))?(a):(b))
#define min(a,b) (((a)<(b))?(a):(b))

#define MAX_BRIGHTNESS 255
#define FADEDIV 2

// pixel base type
// Use int instead `unsigned char' so that we can
// store negative values.
typedef int pixel_t;

__managed__ int offset[16]; // offsets to circle pixels

#define  SIZE (16)

// initializes global array of offsets of neighborhood pixels in circle of radius 3
void makeOffsets(int offset[], const int row_stride)
{
    offset[0] = 0 + row_stride * 3;
    offset[1] = 1 + row_stride * 3;
    offset[2] = 2 + row_stride * 2;
    offset[3] = 3 + row_stride * 1;
    offset[4] = 3 + row_stride * 0;
    offset[5] = 3 + row_stride * -1;
    offset[6] = 2 + row_stride * -2;
    offset[7] = 1 + row_stride * -3;
    offset[8] = 0 + row_stride * -3;
    offset[9] = -1 + row_stride * -3;
    offset[10] = -2 + row_stride * -2;
    offset[11] = -3 + row_stride * -1;
    offset[12] = -3 + row_stride * 0;
    offset[13] = -3 + row_stride * 1;
    offset[14] = -2 + row_stride * 2;
    offset[15] = -1 + row_stride * 3;
}

#define DARKER (-1)
#define SIMILAR 0
#define BRIGHTER 1

// detects if pixel pointed to by h_ipixel is a FAST feature:
//   has at least th_count consecutive neighbours (along circle radius 3)
//   that are darker or brighter by th_diff
__host__ __device__ int fastCorner(const pixel_t *h_ipixel,
               const int w, const int h,
               const int th_count, // min count to detect corners size
               const int th_diff   // threshold diff to count
)
{
    int consec = 0;
    int dk_consec = 0, br_consec = 0;
    int dk_begin = 0, dk_begin_count = 0;
    int br_begin = 0, br_begin_count = 0;

    pixel_t pix_val = *h_ipixel;
    int intensity = SIMILAR;

    int p;
    for (p = 0; p < 16; p++)
    {
        if (h_ipixel[offset[p]] < pix_val - th_diff)
        { // Darker neighbor
            if (p == 0)
            {
                dk_begin = 1;
            }
            if (intensity == DARKER)
            {
                consec++;
            }
            else
            {
                if (intensity == BRIGHTER && consec > br_consec)
                {
                    if (br_begin == 1)
                    {
                        br_begin_count = consec;
                        br_begin = 0;
                    }
                    br_consec = consec;
                }
                consec = 1;
            }
            intensity = DARKER;
        }

        else if (h_ipixel[offset[p]] > pix_val + th_diff)
        { // Brighter neighbor
            if (p == 0)
            {
                br_begin = 1;
            }
            if (intensity == BRIGHTER)
            {
                consec++;
            }
            else
            {
                if (intensity == DARKER && consec > dk_consec)
                {
                    if (dk_begin == 1)
                    {
                        dk_begin_count = consec;
                        dk_begin = 0;
                    }
                    dk_consec = consec;
                }
                consec = 1;
            }
            intensity = BRIGHTER;
        }
        else
        { // Similar Neighbor
            if (intensity == DARKER && consec > dk_consec)
            {
                if (dk_begin == 1)
                {
                    dk_begin_count = consec;
                    dk_begin = 0;
                }
                dk_consec = consec;
            }
            if (intensity == BRIGHTER && consec > br_consec)
            {
                if (br_begin == 1)
                {
                    br_begin_count = consec;
                    br_begin = 0;
                }
                br_consec = consec;
            }
            consec = 0;
            intensity = SIMILAR;
        }
    }

    if (intensity == DARKER)
    {
        if (dk_begin_count)
        { // merge consecutive pixels
            if (consec + dk_begin_count > dk_consec)
                dk_consec = consec + dk_begin_count;
        }
        else if (consec > dk_consec)
        {
            dk_consec = consec;
        }
    }

    if (intensity == BRIGHTER)
    {
        if (br_begin_count)
        { // merge consecutive pixels
            if (consec + br_begin_count > br_consec)
                br_consec = consec + br_begin_count;
        }
        else if (consec > br_consec)
        {
            br_consec = consec;
        }
    }
    if (dk_consec >= th_count || br_consec >= th_count)
    {
        return 1;
    }
    return 0;
}

// returns the score of pixel pointed to by h_ipixel
__host__ __device__ int fastScore(const pixel_t *h_ipixel,
              const int w, const int h,
              const int th_count)
{
    int scoremin = 1;
    int scoremax = max(MAX_BRIGHTNESS-*h_ipixel,*h_ipixel);

    while (scoremax - scoremin > 1)
    {
        if (fastCorner(h_ipixel, w, h, th_count, (scoremin + scoremax) / 2))
        {
            scoremin = (scoremin + scoremax) / 2;
        }
        else {
            scoremax = (scoremin + scoremax) / 2;
        }
    }

    return scoremin;
}


// FAST non-maximum suppression
void nonMaximumSupression(const pixel_t *in, const pixel_t *corners,
                          pixel_t *nms,
                          const int w, const int h, const int th_count)
{
    int count = 0;

    int *corner_score = (int *) malloc(w*h * sizeof(int));

    // determine score of each corner
    for (int i = 0; i < h; i++)
    {
        for (int j = 0; j < w; j++)
        {
            const int c = i * w + j;
            if (corners[c] == MAX_BRIGHTNESS)
            {
                  corner_score[c] = fastScore(in + c, w, h, th_count);
            }
            else {
                    
                  corner_score[c] = 0;
            }
        }
    }

    // keep only corner with local maximum score
    for (int i = 0; i < h; i++)
    {
        for (int j = 0; j < w; j++)
        {
            const int c = i * w + j;
            if (corners[c] != MAX_BRIGHTNESS)
            {
                continue;
            }

            int score_c = corner_score[c];
            int score_n;

            // check all neighbors
            for(int ni = max(0,i-1); ni < min(h,i+2); ni++) {
                for(int nj = max(0,j-1); nj < min(w,j+2); nj++) {
                    if(ni == i && nj == j) continue;

                    int nc = ni * w + nj; 
                    score_n = corner_score[nc];
                    if (score_n >= score_c) {
                        nms[c] = 0;
                        goto next;
                    }
                }
            }

            nms[c] = MAX_BRIGHTNESS;
            count++;
            next: continue;
        }
    }

    free(corner_score);

    //printf("nonmax features %d\n", count);
}

__global__ void cornerScore_Maximum_supression_CUDA(pixel_t *in, pixel_t *corners,
                                            pixel_t *corner_score,
                                            const int w, const int h, const int th_count)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < w && j < h)
    {
        int c = j * w + i;
        if (corners[c] == MAX_BRIGHTNESS)
        {
            corner_score[c] = fastScore(in + c, w, h, th_count);
        }
        else {
            corner_score[c] = 0;
        }
    }
}
// FAST non-maximum suppression - CUDA
__global__ void nonMaximumSupression_CUDA(pixel_t *in, pixel_t *corners,
                                            pixel_t *nms, pixel_t *corner_score,
                                            const int w, const int h, const int th_count)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < w && j < h)
    {
        int c = j * w + i;
        if (corners[c] == MAX_BRIGHTNESS)
        {
            int score_c = corner_score[c];
            int score_n;

            // check all neighbors
            for (int ni = max(0, j - 1); ni < min(h, j + 2); ni++) {
                for (int nj = max(0, i - 1); nj < min(w, i + 2); nj++) {
                    if (ni == j && nj == i) continue;

                    int nc = ni * w + nj;
                    score_n = corner_score[nc];
                    if (score_n >= score_c) {
                        nms[c] = 0;
                        goto next;
                    }
                }
            }

            nms[c] = MAX_BRIGHTNESS;
            next: ;
        }
    }
}

// detects all FAST corners in image h_idata and marks them with MAX_BRIGHTNESS in h_odata
void fastDetectCorners(const pixel_t *h_idata,
                       const int w, const int h,
                       const int th_count, // min count to detect corners size
                       const int th_diff,  // threshold diff to count
                       pixel_t *h_odata)
{
    int i, j, count = 0;

    for (i = 3; i < h - 3; i++) // height image
    {
        for (j = 3; j < w - 3; j++) // width image
        {
            if (fastCorner(h_idata + i * w + j, w, h, th_count, th_diff))
            {
                h_odata[i * w + j] = MAX_BRIGHTNESS;
                count++;
            }
        }
    }
    //printf("detected %d features\n", count);
}
// detects all FAST corners in image h_idata and marks them with MAX_BRIGHTNESS in h_odata - CUDA
__global__ void fastDetectCorners_CUDA(const pixel_t *h_idata,
                       const int w, const int h,
                       const int th_count, // min count to detect corners size
                       const int th_diff,  // threshold diff to count
                       pixel_t *h_odata)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = row * w + col;

    if (col >= 3 && col < w - 3 && row >= 3 && row < h - 3)
    {
        if (fastCorner(h_idata + idx, w, h, th_count, th_diff))
        {
            h_odata[idx] = MAX_BRIGHTNESS;
        }
    }
}

__global__ void faded_devider_CUDA(const pixel_t *h_idata, pixel_t *h_odata,const int w,const int h)
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < h && j < w){
        int idx = i * w + j;
        if(h_odata[idx]!=MAX_BRIGHTNESS) {
                    h_odata[idx] = h_idata[idx] / FADEDIV; // to obtain a faded background image
        }
    }
}

// fast detector code to run on the host
void fastDetectorHost(const pixel_t *h_idata, const int w, const int h,
                      const int th_count, // min count to detect corners size
                      const int th_diff,  // threshold diff to count
                      const bool nonmaxflag,
                      pixel_t *h_odata)
{
    int i, j; // indexes in image

    // initialize h_odata to zero
    memset(h_odata, 0, h*w*sizeof(pixel_t));

    makeOffsets(offset, w);

    // corner detection
    fastDetectCorners(h_idata, w, h, th_count, th_diff, h_odata);

    if (nonmaxflag)
    {
        pixel_t *aux = (pixel_t *)malloc(w * h * sizeof(pixel_t));

        memcpy(aux, h_odata, w * h * sizeof(pixel_t));

        //non max supression
        nonMaximumSupression(h_idata, aux, h_odata, w, h, th_count);

        free(aux);
    }

    //add faded original image as background
    for (i = 0; i < h; i++) // height image
    {
        for (j = 0; j < w; j++) // width image
        {
            if(h_odata[i * w + j]!=MAX_BRIGHTNESS) {
                h_odata[i * w + j] = h_idata[i * w + j] / FADEDIV; // to obtain a faded background image
            }
        }
    }
}

// fast detector code using GPU
void fastDetectorDevice(const pixel_t *h_idata, const int w, const int h,
                        const int th_count, // min count to detect corners size
                        const int th_diff,  // threshold diff to count
                        const bool nonmaxflag,
                        pixel_t *h_odata)
{
    //int i, j; // indexes in image
    pixel_t *cuda_h_idata,*cuda_h_odata;
    int size_all = w*h*sizeof(pixel_t);

    // initialize h_odata to zero
    memset(h_odata, 0, size_all);

    makeOffsets(offset, w);

    hipMalloc((void **)&cuda_h_idata, size_all);
    hipMemcpy(cuda_h_idata,h_idata, size_all,hipMemcpyHostToDevice);
    hipMalloc((void **)&cuda_h_odata, size_all);
    hipMemcpy(cuda_h_odata,h_odata, size_all,hipMemcpyHostToDevice);

    const dim3 block_size(32,32);
    int bl_x = (w+block_size.x-1)/block_size.x;
    int bl_y = (h+block_size.y-1)/block_size.y;
    const dim3 grid_size(bl_x,bl_y);

    fastDetectCorners_CUDA<<<grid_size,block_size>>>(cuda_h_idata, w, h, th_count, th_diff, cuda_h_odata);
    //hipMemcpy(h_odata,cuda_h_odata,size_all,hipMemcpyDeviceToHost);
    if (nonmaxflag)
    {
        //pixel_t *aux = (pixel_t *)malloc(w * h * sizeof(pixel_t));

        //memcpy(aux, h_odata, w * h * sizeof(pixel_t));

        pixel_t *cuda_aux;
        pixel_t *corner_score;
        hipMalloc((void**)&cuda_aux,size_all);
        hipMemcpy(cuda_aux,cuda_h_odata,size_all,hipMemcpyDeviceToDevice);

        hipMalloc((void**)&corner_score,size_all);
        //non max supression - CUDA
        cornerScore_Maximum_supression_CUDA<<<grid_size,block_size>>>(cuda_h_idata, cuda_aux,corner_score, w, h, th_count);
        //hipDeviceSynchronize(); //este e obrigatorio
        nonMaximumSupression_CUDA<<<grid_size,block_size>>>(cuda_h_idata, cuda_aux, cuda_h_odata,corner_score, w, h, th_count);
        //hipDeviceSynchronize();
        //nonMaximumSuppressionKernel<<<grid_size,block_size>>>(cuda_h_idata, cuda_aux, cuda_h_odata, w, h, th_count);
        // o realizado no host
        //nonMaximumSupression(h_idata, aux, h_odata, w, h, th_count);
        //
        //free(aux);
        hipFree(corner_score);
        hipFree(cuda_aux);
    }
    //hipDeviceSynchronize();
    faded_devider_CUDA<<<grid_size,block_size>>>(cuda_h_idata, cuda_h_odata, w, h);
    hipMemcpy(h_odata,cuda_h_odata,size_all,hipMemcpyDeviceToHost);

    /*
    // para testar se contou bem
    int count=0;
    hipMemcpy(h_odata,cuda_h_odata,size_all,hipMemcpyDeviceToHost);
    for (int i = 0; i < w * h; i++)
    {
        if (h_odata[i] == MAX_BRIGHTNESS)
        {
            count++;
        }
    }
    //printf("CUDA - detected %d features\n", count);
    //
    */
    

    //add faded original image as background

    /*
    for (i = 0; i < h; i++) // height image
    {
        for (j = 0; j < w; j++) // width image
        {
            if(h_odata[i * w + j]!=MAX_BRIGHTNESS) {
                h_odata[i * w + j] = h_idata[i * w + j] / FADEDIV; // to obtain a faded background image
            }
        }
    }
    */
    hipFree(cuda_h_idata);
    hipFree(cuda_h_odata);
}

// print command line format
void usage(char *command) 
{
    printf("Usage: %s [-h] [-d device] [-i inputfile] [-o outputfile] [-r referenceFile] [-c th_count] [-t th_diff] [-m]\n", command);
}

// main
int main( int argc, char** argv) 
{

    // default command line options
    int deviceId = 0; // o 0 acaba por ser melhor mas n esta a funcionar
    char *fileIn        = (char *)"house.pgm",
         *fileOut       = (char *)"resultCuda.pgm",
         *referenceOut  = (char *)"referenceCuda.pgm";
    unsigned int th_count = 9, th_diff = 50, nonmaxflag = 0;

    // parse command line arguments
    int opt;
    while ((opt = getopt(argc, argv, "d:i:o:r:c:t:mh")) != -1)
    {
        switch(opt)
        {

            case 'd':
                if(sscanf(optarg,"%d",&deviceId)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;

            case 'i':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }

                fileIn = strdup(optarg);
                break;
            case 'o':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                fileOut = strdup(optarg);
                break;
            case 'r':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                referenceOut = strdup(optarg);
                break;
            case 'c':
                if (strlen(optarg) == 0 || sscanf(optarg, "%d", &th_count) != 1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 't':
                if (strlen(optarg) == 0 || sscanf(optarg, "%d", &th_diff) != 1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 'm':
                nonmaxflag = 1;
                break;
            case 'h':
                usage(argv[0]);
                exit(0);
                break;

        }
    }

    // select cuda device
    checkCudaErrors( hipSetDevice( deviceId ) );
    
    // create events to measure host fast detector time and device fast detector time

    hipEvent_t startH, stopH, startD, stopD;
    checkCudaErrors( hipEventCreate(&startH) );
    checkCudaErrors( hipEventCreate(&stopH)  );
    checkCudaErrors( hipEventCreate(&startD) );
    checkCudaErrors( hipEventCreate(&stopD)  );

    // allocate host memory
    pixel_t * h_idata=NULL;
    unsigned int h,w;

    //load pgm
    if (sdkLoadPGM<pixel_t>(fileIn, &h_idata, &w, &h) != true) {
        printf("Failed to load image file: %s\n", fileIn);
        exit(1);
    }

    // allocate mem for the result on host side
    pixel_t * h_odata   = (pixel_t *) malloc( h*w*sizeof(pixel_t));
    pixel_t * reference = (pixel_t *) malloc( h*w*sizeof(pixel_t));
 
    // detect corners at host
    //printf("start host\n");
    checkCudaErrors( hipEventRecord( startH, 0 ) );
    fastDetectorHost(h_idata, w, h, th_count, th_diff, nonmaxflag, reference);
    //printf("finished host\n");
    checkCudaErrors( hipEventRecord( stopH, 0 ) ); 
    //printf("check errors record\n");
    checkCudaErrors( hipEventSynchronize( stopH ) );
    //printf("cuda synchronize\n");
    // detect corners at GPU
    checkCudaErrors( hipEventRecord( startD, 0 ) );
    //printf("event start\n");
    fastDetectorDevice(h_idata, w, h, th_count, th_diff, nonmaxflag, h_odata);
    checkCudaErrors( hipEventRecord( stopD, 0 ) ); 
    checkCudaErrors( hipEventSynchronize( stopD ) );
    
    // check if kernel execution generated and error
    getLastCudaError("Kernel execution failed");

    float timeH, timeD;
    checkCudaErrors( hipEventElapsedTime( &timeH, startH, stopH ) );
    printf( "Host processing time: %f (ms)\n", timeH);
    checkCudaErrors( hipEventElapsedTime( &timeD, startD, stopD ) );
    printf( "Device processing time: %f (ms)\n", timeD);

    // save output images
    if (sdkSavePGM<pixel_t>(referenceOut, reference, w, h) != true) {
        printf("Failed to save image file: %s\n", referenceOut);
        exit(1);
    }
    if (sdkSavePGM<pixel_t>(fileOut, h_odata, w, h) != true) {
        printf("Failed to save image file: %s\n", fileOut);
        exit(1);
    }

    // cleanup memory
    free( h_idata);
    free( h_odata);
    free( reference);

    checkCudaErrors( hipDeviceReset() );
}
